#include "hip/hip_runtime.h"
// #include "dw_cuda_soa.h"
#include "su3.h"
#include "macros.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>


pauli_soa* create_pauli_soa(int vol)
{
    pauli_soa* obj = (pauli_soa*) malloc(sizeof(pauli_soa));

    (*obj).l1 = (float*) malloc(36 * (vol/2) * sizeof(float));
    (*obj).l2 = (float*) malloc(36 * (vol/2) * sizeof(float));
    (*obj).r1 = (float*) malloc(36 * (vol/2) * sizeof(float));
    (*obj).r2 = (float*) malloc(36 * (vol/2) * sizeof(float));

    return obj;
}

void destroy_pauli_soa(pauli_soa* obj)
{
    free((*obj).l1);
    free((*obj).l2);
    free((*obj).r1);
    free((*obj).r2);

    free(obj);
}

spinor_soa* create_spinor_soa(int vol)
{
    spinor_soa* obj = (spinor_soa*) malloc(sizeof(spinor_soa));

    (*obj).c1.c1.re = (float*) malloc(vol * sizeof(float));
    (*obj).c1.c1.im = (float*) malloc(vol * sizeof(float));
    (*obj).c1.c2.re = (float*) malloc(vol * sizeof(float));
    (*obj).c1.c2.im = (float*) malloc(vol * sizeof(float));
    (*obj).c1.c3.re = (float*) malloc(vol * sizeof(float));
    (*obj).c1.c3.im = (float*) malloc(vol * sizeof(float));
    (*obj).c2.c1.re = (float*) malloc(vol * sizeof(float));
    (*obj).c2.c1.im = (float*) malloc(vol * sizeof(float));
    (*obj).c2.c2.re = (float*) malloc(vol * sizeof(float));
    (*obj).c2.c2.im = (float*) malloc(vol * sizeof(float));
    (*obj).c2.c3.re = (float*) malloc(vol * sizeof(float));
    (*obj).c2.c3.im = (float*) malloc(vol * sizeof(float));
    (*obj).c3.c1.re = (float*) malloc(vol * sizeof(float));
    (*obj).c3.c1.im = (float*) malloc(vol * sizeof(float));
    (*obj).c3.c2.re = (float*) malloc(vol * sizeof(float));
    (*obj).c3.c2.im = (float*) malloc(vol * sizeof(float));
    (*obj).c3.c3.re = (float*) malloc(vol * sizeof(float));
    (*obj).c3.c3.im = (float*) malloc(vol * sizeof(float));
    (*obj).c4.c1.re = (float*) malloc(vol * sizeof(float));
    (*obj).c4.c1.im = (float*) malloc(vol * sizeof(float));
    (*obj).c4.c2.re = (float*) malloc(vol * sizeof(float));
    (*obj).c4.c2.im = (float*) malloc(vol * sizeof(float));
    (*obj).c4.c3.re = (float*) malloc(vol * sizeof(float));
    (*obj).c4.c3.im = (float*) malloc(vol * sizeof(float));

    return obj;
}

void destroy_spinor_soa(spinor_soa* obj)
{
    free((*obj).c1.c1.re);
    free((*obj).c1.c1.im);
    free((*obj).c1.c2.re);
    free((*obj).c1.c2.im);
    free((*obj).c1.c3.re);
    free((*obj).c1.c3.im);
    free((*obj).c2.c1.re);
    free((*obj).c2.c1.im);
    free((*obj).c2.c2.re);
    free((*obj).c2.c2.im);
    free((*obj).c2.c3.re);
    free((*obj).c2.c3.im);
    free((*obj).c3.c1.re);
    free((*obj).c3.c1.im);
    free((*obj).c3.c2.re);
    free((*obj).c3.c2.im);
    free((*obj).c3.c3.re);
    free((*obj).c3.c3.im);
    free((*obj).c4.c1.re);
    free((*obj).c4.c1.im);
    free((*obj).c4.c2.re);
    free((*obj).c4.c2.im);
    free((*obj).c4.c3.re);
    free((*obj).c4.c3.im);

    free(obj);
}

void copy_pauli_aos2soa(pauli* m, pauli_soa* m_soa, int vol)
{
    int i, j, idx;

    idx = 0;
    for (i = 0; i < vol; i += 2) {
        for (j = 0; j < 36; ++j) {
            (*m_soa).l1[j*(vol/2) + idx] = (*(m+i)).u[j];
            (*m_soa).r1[j*(vol/2) + idx] = (*(m+vol+i)).u[j];
        }
        idx++;
    }

    idx = 0;
    for (i = 1; i < vol; i += 2) {
        for (j = 0; j < 36; ++j) {
            (*m_soa).l2[j*(vol/2) + idx] = (*(m+i)).u[j];
            (*m_soa).r2[j*(vol/2) + idx] = (*(m+vol+i)).u[j];
        }
        idx++;
    }
}

void copy_pauli_soa2aos(pauli_soa* m_soa, pauli* m, int vol)
{
    int i, j, idx;

    idx = 0;
    for (i = 0; i < vol; i += 2) {
        for (j = 0; j < 36; ++j) {
            (*(m+i)).u[j]     = (*m_soa).l1[j*(vol/2) + idx];
            (*(m+vol+i)).u[j] = (*m_soa).r1[j*(vol/2) + idx];
        }
        idx++;
    }

    idx = 0;
    for (i = 1; i < vol; i += 2) {
        for (j = 0; j < 36; ++j) {
            (*(m+i)).u[j]     = (*m_soa).l2[j*(vol/2) + idx];
            (*(m+vol+i)).u[j] = (*m_soa).r2[j*(vol/2) + idx];
        }
        idx++;
    }
}

void copy_spinor_aos2soa(spinor* s, spinor_soa* s_soa, int vol)
{
    for (int i = 0; i < vol; ++i) {
        (*s_soa).c1.c1.re[i] = (*(s+i)).c1.c1.re;
        (*s_soa).c1.c1.im[i] = (*(s+i)).c1.c1.im;
        (*s_soa).c1.c2.re[i] = (*(s+i)).c1.c2.re;
        (*s_soa).c1.c2.im[i] = (*(s+i)).c1.c2.im;
        (*s_soa).c1.c3.re[i] = (*(s+i)).c1.c3.re;
        (*s_soa).c1.c3.im[i] = (*(s+i)).c1.c3.im;
        (*s_soa).c2.c1.re[i] = (*(s+i)).c2.c1.re;
        (*s_soa).c2.c1.im[i] = (*(s+i)).c2.c1.im;
        (*s_soa).c2.c2.re[i] = (*(s+i)).c2.c2.re;
        (*s_soa).c2.c2.im[i] = (*(s+i)).c2.c2.im;
        (*s_soa).c2.c3.re[i] = (*(s+i)).c2.c3.re;
        (*s_soa).c2.c3.im[i] = (*(s+i)).c2.c3.im;
        (*s_soa).c3.c1.re[i] = (*(s+i)).c3.c1.re;
        (*s_soa).c3.c1.im[i] = (*(s+i)).c3.c1.im;
        (*s_soa).c3.c2.re[i] = (*(s+i)).c3.c2.re;
        (*s_soa).c3.c2.im[i] = (*(s+i)).c3.c2.im;
        (*s_soa).c3.c3.re[i] = (*(s+i)).c3.c3.re;
        (*s_soa).c3.c3.im[i] = (*(s+i)).c3.c3.im;
        (*s_soa).c4.c1.re[i] = (*(s+i)).c4.c1.re;
        (*s_soa).c4.c1.im[i] = (*(s+i)).c4.c1.im;
        (*s_soa).c4.c2.re[i] = (*(s+i)).c4.c2.re;
        (*s_soa).c4.c2.im[i] = (*(s+i)).c4.c2.im;
        (*s_soa).c4.c3.re[i] = (*(s+i)).c4.c3.re;
        (*s_soa).c4.c3.im[i] = (*(s+i)).c4.c3.im;
    }
}

void copy_spinor_soa2aos(spinor_soa* s_soa, spinor* s, int vol)
{
    for (int i = 0; i < vol; ++i) {
        (*(s+i)).c1.c1.re = (*s_soa).c1.c1.re[i];
        (*(s+i)).c1.c1.im = (*s_soa).c1.c1.im[i];
        (*(s+i)).c1.c2.re = (*s_soa).c1.c2.re[i];
        (*(s+i)).c1.c2.im = (*s_soa).c1.c2.im[i];
        (*(s+i)).c1.c3.re = (*s_soa).c1.c3.re[i];
        (*(s+i)).c1.c3.im = (*s_soa).c1.c3.im[i];
        (*(s+i)).c2.c1.re = (*s_soa).c2.c1.re[i];
        (*(s+i)).c2.c1.im = (*s_soa).c2.c1.im[i];
        (*(s+i)).c2.c2.re = (*s_soa).c2.c2.re[i];
        (*(s+i)).c2.c2.im = (*s_soa).c2.c2.im[i];
        (*(s+i)).c2.c3.re = (*s_soa).c2.c3.re[i];
        (*(s+i)).c2.c3.im = (*s_soa).c2.c3.im[i];
        (*(s+i)).c3.c1.re = (*s_soa).c3.c1.re[i];
        (*(s+i)).c3.c1.im = (*s_soa).c3.c1.im[i];
        (*(s+i)).c3.c2.re = (*s_soa).c3.c2.re[i];
        (*(s+i)).c3.c2.im = (*s_soa).c3.c2.im[i];
        (*(s+i)).c3.c3.re = (*s_soa).c3.c3.re[i];
        (*(s+i)).c3.c3.im = (*s_soa).c3.c3.im[i];
        (*(s+i)).c4.c1.re = (*s_soa).c4.c1.re[i];
        (*(s+i)).c4.c1.im = (*s_soa).c4.c1.im[i];
        (*(s+i)).c4.c2.re = (*s_soa).c4.c2.re[i];
        (*(s+i)).c4.c2.im = (*s_soa).c4.c2.im[i];
        (*(s+i)).c4.c3.re = (*s_soa).c4.c3.re[i];
        (*(s+i)).c4.c3.im = (*s_soa).c4.c3.im[i];
    }
}

__device__
static void mul_pauli(int idx, int sidx, int halfvol, float mu,
                      spinor_soa const *s, spinor_soa *r,
                      float const *m1, float const *m2)
{
    float u[36];
    weyl sloc;

    sloc.c1.c1.re = (*s).c1.c1.re[sidx];
    sloc.c1.c1.im = (*s).c1.c1.im[sidx];
    sloc.c1.c2.re = (*s).c1.c2.re[sidx];
    sloc.c1.c2.im = (*s).c1.c2.im[sidx];
    sloc.c1.c3.re = (*s).c1.c3.re[sidx];
    sloc.c1.c3.im = (*s).c1.c3.im[sidx];
    sloc.c2.c1.re = (*s).c2.c1.re[sidx];
    sloc.c2.c1.im = (*s).c2.c1.im[sidx];
    sloc.c2.c2.re = (*s).c2.c2.re[sidx];
    sloc.c2.c2.im = (*s).c2.c2.im[sidx];
    sloc.c2.c3.re = (*s).c2.c3.re[sidx];
    sloc.c2.c3.im = (*s).c2.c3.im[sidx];

    #pragma unroll
    for (int i = 0; i < halfvol; ++i) {
        u[i] = m1[i*halfvol + idx];
    }

    (*r).c1.c1.re[sidx] =
      u[0]  * sloc.c1.c1.re - mu    * sloc.c1.c1.im + u[6]  * sloc.c1.c2.re -
      u[7]  * sloc.c1.c2.im + u[8]  * sloc.c1.c3.re - u[9]  * sloc.c1.c3.im +
      u[10] * sloc.c2.c1.re - u[11] * sloc.c2.c1.im + u[12] * sloc.c2.c2.re -
      u[13] * sloc.c2.c2.im + u[14] * sloc.c2.c3.re - u[15] * sloc.c2.c3.im;

    (*r).c1.c1.im[sidx] =
      u[0]  * sloc.c1.c1.im + mu    * sloc.c1.c1.re + u[6]  * sloc.c1.c2.im +
      u[7]  * sloc.c1.c2.re + u[8]  * sloc.c1.c3.im + u[9]  * sloc.c1.c3.re +
      u[10] * sloc.c2.c1.im + u[11] * sloc.c2.c1.re + u[12] * sloc.c2.c2.im +
      u[13] * sloc.c2.c2.re + u[14] * sloc.c2.c3.im + u[15] * sloc.c2.c3.re;

    (*r).c1.c2.re[sidx] =
      u[6]  * sloc.c1.c1.re + u[7]  * sloc.c1.c1.im + u[1]  * sloc.c1.c2.re -
      mu    * sloc.c1.c2.im + u[16] * sloc.c1.c3.re - u[17] * sloc.c1.c3.im +
      u[18] * sloc.c2.c1.re - u[19] * sloc.c2.c1.im + u[20] * sloc.c2.c2.re -
      u[21] * sloc.c2.c2.im + u[22] * sloc.c2.c3.re - u[23] * sloc.c2.c3.im;

    (*r).c1.c2.im[sidx] =
      u[6]  * sloc.c1.c1.im - u[7]  * sloc.c1.c1.re + u[1]  * sloc.c1.c2.im +
      mu    * sloc.c1.c2.re + u[16] * sloc.c1.c3.im + u[17] * sloc.c1.c3.re +
      u[18] * sloc.c2.c1.im + u[19] * sloc.c2.c1.re + u[20] * sloc.c2.c2.im +
      u[21] * sloc.c2.c2.re + u[22] * sloc.c2.c3.im + u[23] * sloc.c2.c3.re;

    (*r).c1.c3.re[sidx] =
      u[8]  * sloc.c1.c1.re + u[9]  * sloc.c1.c1.im + u[16] * sloc.c1.c2.re +
      u[17] * sloc.c1.c2.im + u[2]  * sloc.c1.c3.re - mu    * sloc.c1.c3.im +
      u[24] * sloc.c2.c1.re - u[25] * sloc.c2.c1.im + u[26] * sloc.c2.c2.re -
      u[27] * sloc.c2.c2.im + u[28] * sloc.c2.c3.re - u[29] * sloc.c2.c3.im;

    (*r).c1.c3.im[sidx] =
      u[8]  * sloc.c1.c1.im - u[9]  * sloc.c1.c1.re + u[16] * sloc.c1.c2.im -
      u[17] * sloc.c1.c2.re + u[2]  * sloc.c1.c3.im + mu    * sloc.c1.c3.re +
      u[24] * sloc.c2.c1.im + u[25] * sloc.c2.c1.re + u[26] * sloc.c2.c2.im +
      u[27] * sloc.c2.c2.re + u[28] * sloc.c2.c3.im + u[29] * sloc.c2.c3.re;

    (*r).c2.c1.re[sidx] =
      u[10] * sloc.c1.c1.re + u[11] * sloc.c1.c1.im + u[18] * sloc.c1.c2.re +
      u[19] * sloc.c1.c2.im + u[24] * sloc.c1.c3.re + u[25] * sloc.c1.c3.im +
      u[3]  * sloc.c2.c1.re - mu * sloc.c2.c1.im    + u[30] * sloc.c2.c2.re -
      u[31] * sloc.c2.c2.im + u[32] * sloc.c2.c3.re - u[33] * sloc.c2.c3.im;

    (*r).c2.c1.im[sidx] =
      u[10] * sloc.c1.c1.im - u[11] * sloc.c1.c1.re + u[18] * sloc.c1.c2.im -
      u[19] * sloc.c1.c2.re + u[24] * sloc.c1.c3.im - u[25] * sloc.c1.c3.re +
      u[3]  * sloc.c2.c1.im + mu    * sloc.c2.c1.re + u[30] * sloc.c2.c2.im +
      u[31] * sloc.c2.c2.re + u[32] * sloc.c2.c3.im + u[33] * sloc.c2.c3.re;

    (*r).c2.c2.re[sidx] =
      u[12] * sloc.c1.c1.re + u[13] * sloc.c1.c1.im + u[20] * sloc.c1.c2.re +
      u[21] * sloc.c1.c2.im + u[26] * sloc.c1.c3.re + u[27] * sloc.c1.c3.im +
      u[30] * sloc.c2.c1.re + u[31] * sloc.c2.c1.im + u[4]  * sloc.c2.c2.re -
      mu    * sloc.c2.c2.im + u[34] * sloc.c2.c3.re - u[35] * sloc.c2.c3.im;

    (*r).c2.c2.im[sidx] =
      u[12] * sloc.c1.c1.im - u[13] * sloc.c1.c1.re + u[20] * sloc.c1.c2.im -
      u[21] * sloc.c1.c2.re + u[26] * sloc.c1.c3.im - u[27] * sloc.c1.c3.re +
      u[30] * sloc.c2.c1.im - u[31] * sloc.c2.c1.re + u[4]  * sloc.c2.c2.im +
      mu    * sloc.c2.c2.re + u[34] * sloc.c2.c3.im + u[35] * sloc.c2.c3.re;

    (*r).c2.c3.re[sidx] =
      u[14] * sloc.c1.c1.re + u[15] * sloc.c1.c1.im + u[22] * sloc.c1.c2.re +
      u[23] * sloc.c1.c2.im + u[28] * sloc.c1.c3.re + u[29] * sloc.c1.c3.im +
      u[32] * sloc.c2.c1.re + u[33] * sloc.c2.c1.im + u[34] * sloc.c2.c2.re +
      u[35] * sloc.c2.c2.im + u[5]  * sloc.c2.c3.re - mu    * sloc.c2.c3.im;

    (*r).c2.c3.im[sidx] =
      u[14] * sloc.c1.c1.im - u[15] * sloc.c1.c1.re + u[22] * sloc.c1.c2.im -
      u[23] * sloc.c1.c2.re + u[28] * sloc.c1.c3.im - u[29] * sloc.c1.c3.re +
      u[32] * sloc.c2.c1.im - u[33] * sloc.c2.c1.re + u[34] * sloc.c2.c2.im -
      u[35] * sloc.c2.c2.re + u[5]  * sloc.c2.c3.im + mu    * sloc.c2.c3.re;


      sloc.c1.c1.re = (*s).c3.c1.re[sidx];
      sloc.c1.c1.im = (*s).c3.c1.im[sidx];
      sloc.c1.c2.re = (*s).c3.c2.re[sidx];
      sloc.c1.c2.im = (*s).c3.c2.im[sidx];
      sloc.c1.c3.re = (*s).c3.c3.re[sidx];
      sloc.c1.c3.im = (*s).c3.c3.im[sidx];
      sloc.c2.c1.re = (*s).c4.c1.re[sidx];
      sloc.c2.c1.im = (*s).c4.c1.im[sidx];
      sloc.c2.c2.re = (*s).c4.c2.re[sidx];
      sloc.c2.c2.im = (*s).c4.c2.im[sidx];
      sloc.c2.c3.re = (*s).c4.c3.re[sidx];
      sloc.c2.c3.im = (*s).c4.c3.im[sidx];

      #pragma unroll
      for (int i = 0; i < halfvol; ++i) {
          u[i] = m2[i*halfvol + idx];
      }

      mu = -mu;

      (*r).c3.c1.re[sidx] =
        u[0]  * sloc.c1.c1.re - mu    * sloc.c1.c1.im + u[6]  * sloc.c1.c2.re -
        u[7]  * sloc.c1.c2.im + u[8]  * sloc.c1.c3.re - u[9]  * sloc.c1.c3.im +
        u[10] * sloc.c2.c1.re - u[11] * sloc.c2.c1.im + u[12] * sloc.c2.c2.re -
        u[13] * sloc.c2.c2.im + u[14] * sloc.c2.c3.re - u[15] * sloc.c2.c3.im;

      (*r).c3.c1.im[sidx] =
        u[0]  * sloc.c1.c1.im + mu    * sloc.c1.c1.re + u[6]  * sloc.c1.c2.im +
        u[7]  * sloc.c1.c2.re + u[8]  * sloc.c1.c3.im + u[9]  * sloc.c1.c3.re +
        u[10] * sloc.c2.c1.im + u[11] * sloc.c2.c1.re + u[12] * sloc.c2.c2.im +
        u[13] * sloc.c2.c2.re + u[14] * sloc.c2.c3.im + u[15] * sloc.c2.c3.re;

      (*r).c3.c2.re[sidx] =
        u[6]  * sloc.c1.c1.re + u[7]  * sloc.c1.c1.im + u[1]  * sloc.c1.c2.re -
        mu    * sloc.c1.c2.im + u[16] * sloc.c1.c3.re - u[17] * sloc.c1.c3.im +
        u[18] * sloc.c2.c1.re - u[19] * sloc.c2.c1.im + u[20] * sloc.c2.c2.re -
        u[21] * sloc.c2.c2.im + u[22] * sloc.c2.c3.re - u[23] * sloc.c2.c3.im;

      (*r).c3.c2.im[sidx] =
        u[6]  * sloc.c1.c1.im - u[7]  * sloc.c1.c1.re + u[1]  * sloc.c1.c2.im +
        mu    * sloc.c1.c2.re + u[16] * sloc.c1.c3.im + u[17] * sloc.c1.c3.re +
        u[18] * sloc.c2.c1.im + u[19] * sloc.c2.c1.re + u[20] * sloc.c2.c2.im +
        u[21] * sloc.c2.c2.re + u[22] * sloc.c2.c3.im + u[23] * sloc.c2.c3.re;

      (*r).c3.c3.re[sidx] =
        u[8]  * sloc.c1.c1.re + u[9]  * sloc.c1.c1.im + u[16] * sloc.c1.c2.re +
        u[17] * sloc.c1.c2.im + u[2]  * sloc.c1.c3.re - mu    * sloc.c1.c3.im +
        u[24] * sloc.c2.c1.re - u[25] * sloc.c2.c1.im + u[26] * sloc.c2.c2.re -
        u[27] * sloc.c2.c2.im + u[28] * sloc.c2.c3.re - u[29] * sloc.c2.c3.im;

      (*r).c3.c3.im[sidx] =
        u[8]  * sloc.c1.c1.im - u[9]  * sloc.c1.c1.re + u[16] * sloc.c1.c2.im -
        u[17] * sloc.c1.c2.re + u[2]  * sloc.c1.c3.im + mu    * sloc.c1.c3.re +
        u[24] * sloc.c2.c1.im + u[25] * sloc.c2.c1.re + u[26] * sloc.c2.c2.im +
        u[27] * sloc.c2.c2.re + u[28] * sloc.c2.c3.im + u[29] * sloc.c2.c3.re;

      (*r).c4.c1.re[sidx] =
        u[10] * sloc.c1.c1.re + u[11] * sloc.c1.c1.im + u[18] * sloc.c1.c2.re +
        u[19] * sloc.c1.c2.im + u[24] * sloc.c1.c3.re + u[25] * sloc.c1.c3.im +
        u[3]  * sloc.c2.c1.re - mu * sloc.c2.c1.im    + u[30] * sloc.c2.c2.re -
        u[31] * sloc.c2.c2.im + u[32] * sloc.c2.c3.re - u[33] * sloc.c2.c3.im;

      (*r).c4.c1.im[sidx] =
        u[10] * sloc.c1.c1.im - u[11] * sloc.c1.c1.re + u[18] * sloc.c1.c2.im -
        u[19] * sloc.c1.c2.re + u[24] * sloc.c1.c3.im - u[25] * sloc.c1.c3.re +
        u[3]  * sloc.c2.c1.im + mu    * sloc.c2.c1.re + u[30] * sloc.c2.c2.im +
        u[31] * sloc.c2.c2.re + u[32] * sloc.c2.c3.im + u[33] * sloc.c2.c3.re;

      (*r).c4.c2.re[sidx] =
        u[12] * sloc.c1.c1.re + u[13] * sloc.c1.c1.im + u[20] * sloc.c1.c2.re +
        u[21] * sloc.c1.c2.im + u[26] * sloc.c1.c3.re + u[27] * sloc.c1.c3.im +
        u[30] * sloc.c2.c1.re + u[31] * sloc.c2.c1.im + u[4]  * sloc.c2.c2.re -
        mu    * sloc.c2.c2.im + u[34] * sloc.c2.c3.re - u[35] * sloc.c2.c3.im;

      (*r).c4.c2.im[sidx] =
        u[12] * sloc.c1.c1.im - u[13] * sloc.c1.c1.re + u[20] * sloc.c1.c2.im -
        u[21] * sloc.c1.c2.re + u[26] * sloc.c1.c3.im - u[27] * sloc.c1.c3.re +
        u[30] * sloc.c2.c1.im - u[31] * sloc.c2.c1.re + u[4]  * sloc.c2.c2.im +
        mu    * sloc.c2.c2.re + u[34] * sloc.c2.c3.im + u[35] * sloc.c2.c3.re;

      (*r).c4.c3.re[sidx] =
        u[14] * sloc.c1.c1.re + u[15] * sloc.c1.c1.im + u[22] * sloc.c1.c2.re +
        u[23] * sloc.c1.c2.im + u[28] * sloc.c1.c3.re + u[29] * sloc.c1.c3.im +
        u[32] * sloc.c2.c1.re + u[33] * sloc.c2.c1.im + u[34] * sloc.c2.c2.re +
        u[35] * sloc.c2.c2.im + u[5]  * sloc.c2.c3.re - mu    * sloc.c2.c3.im;

      (*r).c4.c3.im[sidx] =
        u[14] * sloc.c1.c1.im - u[15] * sloc.c1.c1.re + u[22] * sloc.c1.c2.im -
        u[23] * sloc.c1.c2.re + u[28] * sloc.c1.c3.im - u[29] * sloc.c1.c3.re +
        u[32] * sloc.c2.c1.im - u[33] * sloc.c2.c1.re + u[34] * sloc.c2.c2.im -
        u[35] * sloc.c2.c2.re + u[5]  * sloc.c2.c3.im + mu    * sloc.c2.c3.re;
}

extern "C" __global__
void mulpauli_kernel(int VOLUME, float mu, spinor_soa *s, spinor_soa *r, pauli_soa *m)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= VOLUME/2) return;

    int sidx;
    int halfvol = VOLUME/2;

    sidx = idx;
    mul_pauli(idx, sidx, halfvol, mu, s, r, (*m).l1, (*m).l2);

    sidx = halfvol + idx;
    mul_pauli(idx, sidx, halfvol, mu, s, r, (*m).r1, (*m).r2);
}

extern "C"
void Dw_cuda_SoA(int VOLUME, su3 *u, spinor *s, spinor *r, pauli *m, int *piup, int *pidn)
{
    float mu, coe, ceo;
    float gamma_f, one_over_gammaf;

    mu = 0.0f;
    coe = -0.5f;
    ceo = -0.5f;

    gamma_f = 1.0f;
    one_over_gammaf = 1.0f;


    // Create structure of arrays
    pauli_soa *m_soa = create_pauli_soa(VOLUME);
    spinor_soa *s_soa = create_spinor_soa(VOLUME);
    spinor_soa *r_soa = create_spinor_soa(VOLUME);

    // Copy data from AoS to SoA
    copy_pauli_aos2soa(m, m_soa, VOLUME);
    copy_spinor_aos2soa(s, s_soa, VOLUME);
    // copy_spinor_aos2soa(r, r_soa, VOLUME);

    // Allocate memory space on device
    pauli_soa *d_m_soa;
    spinor_soa *d_s_soa, *d_r_soa;

    printf("Before hipMalloc\n");
    hipMalloc((void **)&d_m_soa, sizeof(*d_m_soa));
    printf("Before hipMalloc m interior\n");
    hipMalloc((void **)&((*d_m_soa).l1), 36 * (VOLUME/2) * sizeof(float));
    hipMalloc((void **)&((*d_m_soa).l2), 36 * (VOLUME/2) * sizeof(float));
    hipMalloc((void **)&((*d_m_soa).r1), 36 * (VOLUME/2) * sizeof(float));
    hipMalloc((void **)&((*d_m_soa).r2), 36 * (VOLUME/2) * sizeof(float));

    printf("Before hipMalloc for spinors\n");
    hipMalloc((void **)&d_s_soa, sizeof(*d_s_soa));
    hipMalloc((void **)&((*d_s_soa).c1.c1.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c1.c1.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c1.c2.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c1.c2.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c1.c3.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c1.c3.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c2.c1.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c2.c1.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c2.c2.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c2.c2.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c2.c3.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c2.c3.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c3.c1.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c3.c1.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c3.c2.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c3.c2.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c3.c3.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c3.c3.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c4.c1.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c4.c1.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c4.c2.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c4.c2.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c4.c3.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_s_soa).c4.c3.im), VOLUME * sizeof(float));

    hipMalloc((void **)&d_r_soa, sizeof(*d_r_soa));
    hipMalloc((void **)&((*d_r_soa).c1.c1.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c1.c1.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c1.c2.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c1.c2.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c1.c3.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c1.c3.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c2.c1.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c2.c1.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c2.c2.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c2.c2.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c2.c3.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c2.c3.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c3.c1.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c3.c1.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c3.c2.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c3.c2.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c3.c3.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c3.c3.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c4.c1.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c4.c1.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c4.c2.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c4.c2.im), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c4.c3.re), VOLUME * sizeof(float));
    hipMalloc((void **)&((*d_r_soa).c4.c3.im), VOLUME * sizeof(float));


    printf("Before hipMemcpyHostToDevice\n");
    // Copy data from host to device
    hipMemcpy((*d_m_soa).l1, (*m_soa).l1, 36 * (VOLUME/2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_m_soa).l2, (*m_soa).l2, 36 * (VOLUME/2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_m_soa).r1, (*m_soa).r1, 36 * (VOLUME/2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_m_soa).r2, (*m_soa).r2, 36 * (VOLUME/2) * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy((*d_s_soa).c1.c1.re, (*s_soa).c1.c1.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c1.c1.im, (*s_soa).c1.c1.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c1.c2.re, (*s_soa).c1.c2.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c1.c2.im, (*s_soa).c1.c2.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c1.c3.re, (*s_soa).c1.c3.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c1.c3.im, (*s_soa).c1.c3.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c2.c1.re, (*s_soa).c2.c1.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c2.c1.im, (*s_soa).c2.c1.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c2.c2.re, (*s_soa).c2.c2.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c2.c2.im, (*s_soa).c2.c2.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c2.c3.re, (*s_soa).c2.c3.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c2.c3.im, (*s_soa).c2.c3.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c3.c1.re, (*s_soa).c3.c1.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c3.c1.im, (*s_soa).c3.c1.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c3.c2.re, (*s_soa).c3.c2.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c3.c2.im, (*s_soa).c3.c2.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c3.c3.re, (*s_soa).c3.c3.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c3.c3.im, (*s_soa).c3.c3.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c4.c1.re, (*s_soa).c4.c1.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c4.c1.im, (*s_soa).c4.c1.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c4.c2.re, (*s_soa).c4.c2.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c4.c2.im, (*s_soa).c4.c2.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c4.c3.re, (*s_soa).c4.c3.re, VOLUME * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((*d_s_soa).c4.c3.im, (*s_soa).c4.c3.im, VOLUME * sizeof(float), hipMemcpyHostToDevice);


    // Launch kernel on GPU
    int block_size = 128;
    int grid_size = ceil((VOLUME/2.0)/(float)block_size);
    mulpauli_kernel<<<grid_size, block_size>>>(VOLUME, mu, d_s_soa, d_r_soa, d_m_soa);


    printf("Before hipMemcpyDeviceToHost\n");
    // Copy data from device to host
    hipMemcpy((*r_soa).c1.c1.re, (*d_r_soa).c1.c1.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c1.c1.im, (*d_r_soa).c1.c1.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c1.c2.re, (*d_r_soa).c1.c2.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c1.c2.im, (*d_r_soa).c1.c2.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c1.c3.re, (*d_r_soa).c1.c3.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c1.c3.im, (*d_r_soa).c1.c3.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c2.c1.re, (*d_r_soa).c2.c1.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c2.c1.im, (*d_r_soa).c2.c1.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c2.c2.re, (*d_r_soa).c2.c2.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c2.c2.im, (*d_r_soa).c2.c2.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c2.c3.re, (*d_r_soa).c2.c3.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c2.c3.im, (*d_r_soa).c2.c3.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c3.c1.re, (*d_r_soa).c3.c1.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c3.c1.im, (*d_r_soa).c3.c1.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c3.c2.re, (*d_r_soa).c3.c2.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c3.c2.im, (*d_r_soa).c3.c2.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c3.c3.re, (*d_r_soa).c3.c3.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c3.c3.im, (*d_r_soa).c3.c3.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c4.c1.re, (*d_r_soa).c4.c1.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c4.c1.im, (*d_r_soa).c4.c1.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c4.c2.re, (*d_r_soa).c4.c2.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c4.c2.im, (*d_r_soa).c4.c2.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c4.c3.re, (*d_r_soa).c4.c3.re, VOLUME * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((*r_soa).c4.c3.im, (*d_r_soa).c4.c3.im, VOLUME * sizeof(float), hipMemcpyDeviceToHost);


    // Free GPU memory
    // Do that at some point

    printf("Before Convert from SoA to AoS\n");
    // Convert from SoA to AoS
    copy_spinor_soa2aos(r_soa, r, VOLUME);

    // Free SoA
    destroy_pauli_soa(m_soa);
    destroy_spinor_soa(s_soa);
    destroy_spinor_soa(r_soa);


}
